#include "hip/hip_runtime.h"
#include <bits/stdc++.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

using namespace std;

int num_vertices;
int num_edges;
int max_degree;
int *num_edges_per_vertex;
int *adjacency_list;

int* device_m;
int* device_adj_list;
int* device_conflicts;
int* device_temp_conflicts;
int* device_colors;
int* device_new_colors;
bool* device_forbidden;

#define BILLION  1000000000.0

double assign_colors_time = 0;
double detect_conflicts_time = 0;
double total_time = 0;
double memory_ops_time = 0;
int iterations = 0;
struct timespec m_start, m_end;

void printGraph(int n, int m[], int *adj_list) {
    for (int i = 0; i < n; ++i) {
        cout << "Node " << i << " || ";
        for (int j = 0; j < m[i]; ++j) {
            cout << adj_list[i * max_degree + j] << "    ";
        }
        cout << endl;
    }
}

__global__ void assign_init_values(int* conflicts, int num_vertices) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_vertices) return;
    conflicts[i] = i;
}

__global__ void assign_colors_kernel(int num_conflicts, int *conflicts, int maxd, int *m, int *adj_list, int *colors,
                                     int* new_colors, bool* forbidden) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_conflicts) return;

    for (int j = 0; j < maxd + 1; ++j) {
        forbidden[i*(maxd+1) + j] = false;
    }

    int v = conflicts[i];

    for (int j = 0; j < m[v]; ++j) {
        int u = adj_list[v * maxd + j];
        if (colors[u] >= 0)
            forbidden[i * (maxd + 1) + colors[u]] = true;
    }

    for (int j = 0; j < maxd + 1; ++j) {
        if (!forbidden[i * (maxd + 1) + j]) {
            new_colors[v] = j;
            break;
        }
    }
}

__global__ void upsweep(int *data, int N, int twod, int twod1) {
    int i = (blockIdx.x * blockDim.x + threadIdx.x) * twod1;
    if (i + twod1 - 1 < N)
        data[i + twod1 - 1] += data[i + twod - 1];
}

__global__ void downsweep(int *data, int twod, int twod1) {
    int i = (blockIdx.x * blockDim.x + threadIdx.x) * twod1;
    int t = data[i + twod - 1];
    data[i + twod - 1] = data[i + twod1 - 1];
    data[i + twod1 - 1] += t;
}

__global__ void findConflicts(int* conflicts, int* temp_conflicts, int num_vertices) {
    int i = (blockIdx.x * blockDim.x + threadIdx.x);
    if (i >= num_vertices) return;
    if (temp_conflicts[i] < temp_conflicts[i+1]) {
        conflicts[temp_conflicts[i]] = i;
    }
}

__global__ void detectConflictsKernel(int* conflicts, int* adj_list, int* temp_conflicts, int* colors, int* m, int
max_degree, int num_vertices) {
    int i = (blockIdx.x * blockDim.x + threadIdx.x);
    if (i >= num_vertices) return;

    int v = conflicts[i];
    for (int j = 0; j < m[v]; ++j) {
        int u = adj_list[v * max_degree + j];
        if (colors[u] == colors[v] && u < v) {
            temp_conflicts[u] = 1;
            colors[u] = -u;
        }
    }
}

int nextPow2(int N)
{
    unsigned count = 0;

    if (N && !(N & (N - 1)))
        return N;

    while(N != 0)
    {   N>>= 1;
        count += 1;
    }

    return 1 << count;
}

void exclusive_scan(int *device_data, int length) {
    int orig_length = length;

    length = nextPow2(length);

    // upsweep phase.
    for (int twod = 1; twod < length; twod *= 2) {
        int twod1 = twod * 2;
        const int threadsPerBlock = (512 > length / twod1) ? length / twod1 : 512;
        const int blocks = ((length / twod1) + threadsPerBlock - 1) / threadsPerBlock;
        upsweep << < blocks, threadsPerBlock >> > (device_data, orig_length, twod, twod1);
        hipDeviceSynchronize();
    }

    // Setting the last element to zero
    hipMemset(device_data + length - 1, 0, sizeof(int));

    // downsweep phase.
    for (int twod = length / 2; twod >= 1; twod /= 2) {
        int twod1 = twod * 2;
        const int threadsPerBlock = (512 > length / twod1) ? length / twod1 : 512;
        const int blocks = ((length / twod1) + threadsPerBlock - 1) / threadsPerBlock;
        downsweep << < blocks, threadsPerBlock >> > (device_data, twod, twod1);
        hipDeviceSynchronize();
    }
}

void assign_colors(int num_conflicts) {
    clock_gettime(CLOCK_REALTIME, &m_start);
    hipMemcpy(device_colors, device_new_colors, num_vertices * sizeof(int), hipMemcpyDeviceToDevice);
    clock_gettime(CLOCK_REALTIME, &m_end);

    memory_ops_time += (m_end.tv_sec - m_start.tv_sec) +
                       (m_end.tv_nsec - m_start.tv_nsec) / BILLION;

    assign_colors_kernel <<<(num_conflicts+1023)/1024, 1024>>> (num_conflicts, device_conflicts, max_degree, device_m,
            device_adj_list,
            device_colors, device_new_colors, device_forbidden);
    hipDeviceSynchronize();
}

void detect_conflicts(int num_conflicts, int *temp_num_conflicts) {
    clock_gettime(CLOCK_REALTIME, &m_start);
    hipMemset((void*) device_temp_conflicts, 0, (num_vertices+1) * sizeof(int));
    clock_gettime(CLOCK_REALTIME, &m_end);
    memory_ops_time += (m_end.tv_sec - m_start.tv_sec) +
                       (m_end.tv_nsec - m_start.tv_nsec) / BILLION;

    detectConflictsKernel<<<(num_conflicts+1023)/1024, 1024>>> (device_conflicts, device_adj_list, device_temp_conflicts,
            device_new_colors, device_m, max_degree, num_conflicts);
    hipDeviceSynchronize();

    exclusive_scan(device_temp_conflicts, num_vertices + 1);

    hipMemcpy(temp_num_conflicts, device_temp_conflicts + num_vertices, sizeof(int),
               hipMemcpyDeviceToHost);

    findConflicts<<<(num_vertices+1023)/1024, 1024>>> (device_conflicts, device_temp_conflicts, num_vertices);
    hipDeviceSynchronize();
}

int* IPGC() {
    int *colors = (int *) calloc(num_vertices, sizeof(int));
    int num_conflicts = num_vertices;

    assign_init_values<<<(num_vertices+1023)/1024, 1024>>>(device_conflicts, num_vertices);

    int temp_num_conflicts = 0;

    struct timespec start, end, start1, end1;

    clock_gettime(CLOCK_REALTIME, &start);
    while (num_conflicts) {
        iterations++;

        clock_gettime(CLOCK_REALTIME, &start1);
        assign_colors(num_conflicts);
        clock_gettime(CLOCK_REALTIME, &end1);
        assign_colors_time += (end1.tv_sec - start1.tv_sec) +
                              (end1.tv_nsec - start1.tv_nsec) / BILLION;

        clock_gettime(CLOCK_REALTIME, &start1);
        detect_conflicts(num_conflicts, &temp_num_conflicts);
        clock_gettime(CLOCK_REALTIME, &end1);
        detect_conflicts_time += (end1.tv_sec - start1.tv_sec) +
                                 (end1.tv_nsec - start1.tv_nsec) / BILLION;

        num_conflicts = temp_num_conflicts;
        temp_num_conflicts = 0;
    }
    clock_gettime(CLOCK_REALTIME, &end);
    total_time += (end.tv_sec - start.tv_sec) +
                 (end.tv_nsec - start.tv_nsec) / BILLION;

    hipMemcpy(colors, device_new_colors, num_vertices * sizeof(int), hipMemcpyDeviceToHost);
//    for (int i = 0; i < num_vertices; ++i) {
//        cout << "Color of node " << i << " : " << colors[i] << endl;
//    }
    fflush(stdin);
    fflush(stdout);
    return colors;
}

bool checker(int nvertices, int *num_edges, int *colors, int *adjacency_list) {
    bool passed = true;
    for (int i = 0; i < nvertices; ++i) {
        for (int j = 0; j < num_edges[i]; ++j) {
            if (colors[i] == colors[adjacency_list[i * max_degree + j]] || colors[i] < 0 || colors[i] > max_degree +
                                                                                                        1) {
                passed = false;
                cout << "Failed coloring between nodes : " << i << " -- " << adjacency_list[i * max_degree + j];
                fflush(stdin);
                fflush(stdout);
                break;
            }
        }
    }
    return passed;
}

int main(int argc, char *argv[]) {
    char *filename = argv[1];

    ifstream fin(filename);
    fin >> (max_degree);
    fin >> (num_vertices);

    adjacency_list = (int *) malloc(num_vertices * (max_degree) * sizeof(int));
    num_edges_per_vertex = (int *) malloc(num_vertices * sizeof(int ));

    for (int i = 0; i < num_vertices; ++i) {
        fin >> num_edges_per_vertex[i];
        for (int j = 0; j < num_edges_per_vertex[i]; ++j) {
            fin >> adjacency_list[i * max_degree + j];
        }
    }
    fin.close();

    struct timespec start, end;
    clock_gettime(CLOCK_REALTIME, &start);

    hipMalloc((void**) &device_m, num_vertices * sizeof(int));
    hipMalloc((void**) &device_adj_list, max_degree * num_vertices * sizeof(int));
    hipMalloc((void**) &device_temp_conflicts, nextPow2(num_vertices + 1) * sizeof(int));
    hipMalloc((void**) &device_conflicts, (num_vertices + 1) * sizeof(int));
    hipMalloc((void**) &device_colors, num_vertices * sizeof(int));
    hipMalloc((void**) &device_new_colors, num_vertices * sizeof(int));
    hipMalloc((void**) &device_forbidden, (max_degree+1) * num_vertices * sizeof(bool));

    hipMemcpy(device_adj_list, adjacency_list, max_degree * num_vertices * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_m, num_edges_per_vertex, num_vertices * sizeof(int), hipMemcpyHostToDevice);

    clock_gettime(CLOCK_REALTIME, &end);
    total_time += (end.tv_sec - start.tv_sec) +
                 (end.tv_nsec - start.tv_nsec) / BILLION;
    memory_ops_time += (end.tv_sec - start.tv_sec) +
                       (end.tv_nsec - start.tv_nsec) / BILLION;

//	printGraph(nvertices, num_edges, adjacency_list);
    int *colors = IPGC();
    cout << "Total time for coloring : " << total_time * 1000 << " ms" << endl;
    cout << "Time taken for Assign Colors : " << assign_colors_time * 1000 << " ms" << endl;
    cout << "Time taken for Detect Conflicts : " << detect_conflicts_time * 1000 << " ms" << endl;
    cout << "Time taken for Memory operations: " << memory_ops_time * 1000 << " ms" << endl;
    cout << "Iterations taken to converge : " << iterations << endl;
    int max_color = -1;
    for (int i = 0; i < num_vertices; ++i) {
        max_color = max(max_color, colors[i]);
    }
    cout << "Colors used in the coloring : " << max_color + 1 << endl;

    if (checker(num_vertices, num_edges_per_vertex, colors, adjacency_list)) {
        cout << "CORRECT COLORING!!!" << endl;
    } else {
        cout << "INCORRECT COLORING!!!" << endl;
    }
}