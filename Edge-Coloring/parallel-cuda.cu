#include "hip/hip_runtime.h"
#include <bits/stdc++.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

using namespace std;

int max_degree;
int num_vertices;
int num_edges;
int *edges;

int* device_colors;
bool* device_vforbidden;
int* device_edges;
int* device_temp_colors;
bool* device_is_conflict;

__global__ void assign_colors_kernel(int *colors, bool* vforbidden, int max_degree, int num_vertices) {
    int i = (blockIdx.x * blockDim.x + threadIdx.x);

    if (i >= num_vertices) return;
    if (colors[i] != -1)
        return;

    for (int j = 0; j < max_degree + 1; ++j) {
        if (!vforbidden[i * (max_degree + 1) + j]) {
            colors[i] = j;
            break;
        }
    }
}

__global__ void detect_conflicts_kernel(int* edges, int* colors, int* temp_colors, bool* vforbidden,
        bool* is_conflict, int max_degree, int num_edges) {
    int i = (blockIdx.x * blockDim.x + threadIdx.x);

    if (i >= num_edges) return;

    int smaller_vertex, bigger_vertex;
    if (edges[2*i] > edges[2*i+1]) {
        bigger_vertex = edges[2*i];
        smaller_vertex = edges[2*i+1];
    } else {
        bigger_vertex = edges[2*i+1];
        smaller_vertex = edges[2*i];
    }
    if (colors[smaller_vertex] == colors[bigger_vertex]) {
        temp_colors[smaller_vertex] = -1;
        if (*is_conflict == false)
            *is_conflict = true;
    }
    vforbidden[smaller_vertex * (max_degree+1) + colors[bigger_vertex]] = true;
}

void assign_colors() {
    assign_colors_kernel<<<(num_vertices+1023)/1024, 1024>>>(device_colors, device_vforbidden, max_degree,
            num_vertices);
    hipDeviceSynchronize();
}

bool detect_conflicts() {
    bool is_conflict = false;
    hipMemset(device_is_conflict, false, sizeof(bool));

    detect_conflicts_kernel<<<(num_edges + 1023)/1024, 1024>>>(device_edges, device_colors, device_temp_colors,
            device_vforbidden,
            device_is_conflict, max_degree, num_edges);
    hipDeviceSynchronize();

    hipMemcpy(&is_conflict, device_is_conflict, sizeof(bool), hipMemcpyDeviceToHost);
    return is_conflict;
}

int *IPGC() {
    int *colors = (int *) malloc(num_vertices * sizeof(int));
    hipMemset(device_colors, -1, num_vertices * sizeof(int));

    long iter = 0;
    int is_conflict = true;

    while (is_conflict) {
        cout << __LINE__ << endl;
        fflush(stdin);
        fflush(stdout);

        iter++;
        assign_colors();

        hipMemset(device_vforbidden, 0, num_vertices * (max_degree + 1) * sizeof(bool));
        hipMemcpy(device_temp_colors, device_colors, num_vertices * sizeof(int), hipMemcpyDeviceToDevice);

        is_conflict = detect_conflicts();

        hipMemcpy(device_colors, device_temp_colors, num_vertices * sizeof(int), hipMemcpyDeviceToDevice);
    }

    hipMemcpy(colors, device_colors, num_vertices * sizeof(int), hipMemcpyDeviceToHost);
    cout << "Iteration " << iter << endl;
    for (int i = 0; i < num_vertices; ++i) {
        cout << "Color of node " << i << " : " << colors[i] << endl;
    }
    fflush(stdin);
    fflush(stdout);
    return colors;
}

bool checker(int *colors) {
    bool passed = true;
    for (int i = 0; i < num_edges; ++i) {
        if (colors[edges[2*i]] == colors[edges[2*i+1]] ||
        colors[edges[2*i]] < 0 || colors[edges[2*i+1]] < 0 ||
        colors[edges[2*i]] > max_degree + 1 || colors[edges[2*i+1]] > max_degree + 1) {
            passed = false;
        }
    }
    return passed;
}

int main(int argc, char *argv[]) {
    char *filename = argv[1];

    cout << filename << endl;
    ifstream fin(filename);
    fin >> max_degree;
    fin >> (num_vertices);
    fin >> (num_edges);
    cout << max_degree << " : " << num_vertices << " : " << num_edges << endl;
    fflush(stdin);
    fflush(stdout);
    edges = (int *) malloc(2 * num_edges * sizeof(int));

    for (int i = 0; i < num_edges; ++i) {
        fin >> edges[2*i];
        fin >> edges[2*i+1];
        fflush(stdin);
        fflush(stdout);
    }
    fin.close();

    hipMalloc((void**) &device_colors, num_vertices * sizeof(int));
    hipMalloc((void**) &device_vforbidden, num_vertices * (max_degree + 1) * sizeof(bool));
    hipMalloc((void**) &device_edges, 2 * num_edges * sizeof(int));
    hipMalloc((void**) &device_temp_colors, num_vertices * sizeof(int));
    hipMalloc((void**) &device_is_conflict, sizeof(bool));

    hipMemcpy(device_edges, edges, 2 * num_edges * sizeof(int), hipMemcpyHostToDevice);

    int *colors = IPGC();

    cout << "Coloring done!" << endl;
    if (checker(colors)) {
        cout << "CORRECT COLORING!!!" << endl;
    } else {
        cout << "INCORRECT COLORING!!!" << endl;
    }
}